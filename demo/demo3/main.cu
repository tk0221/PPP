
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int * d_out, int * d_in){
	
    int idx = threadIdx.x;
    int f = d_in[idx];
    d_out[idx]=f+f;

    
} //

int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	// generate the input array on the host
	int h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = i;
	}
	int h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	int * d_in;
	int * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	add<<<1, ARRAY_SIZE>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%d", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
